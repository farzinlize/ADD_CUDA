#include "hip/hip_runtime.h"
#include "kernels.cuh"

__device__ void warpReduce(volatile int* sdata, int tid)
{
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
}

#ifdef IN_ARRAY
__global__ void add_kernel_in_array(arguments args)
#else
__global__ void add_kernel(arguments args)
#endif
{
	extern __shared__ int a_s[];
	unsigned int tid_block = threadIdx.x;
	unsigned int tid = (blockDim.x*2) * blockIdx.x + tid_block;
	
	a_s[tid_block] = args.a_in[tid] + args.a_in[tid+blockDim.x];
    __syncthreads();

    for (unsigned int s = blockDim.x/2; s > 32 ; s >>= 1){
		if (tid_block < s)
			a_s[tid_block] = a_s[tid_block] + a_s[tid_block + s];
		__syncthreads();
	}

	if (tid_block<32) warpReduce(a_s, tid_block);

	if (tid_block == 0){
		#ifdef IN_ARRAY
		args.a_in[blockIdx.x] = a_s[0];
		#else
		args.out[blockIdx.x] = a_s[0];
		#endif
	}
}