#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
extern "C"{
    #include "helper_functions.h"
    #include "fuzzy_timing.h"
}
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_kernel(int *a_in)
{
	extern __shared__ int a_s[];
	unsigned int tid_block = threadIdx.x;
	unsigned int tid = (blockDim.x*2) * blockIdx.x + tid_block;
	
	a_s[tid_block] = a_in[tid] + a_in[tid+blockDim.x];
	__syncthreads();

    for (unsigned int s = blockDim.x/2; s > 0 ; s >>= 1)
    {
		if (tid_block < s)
			a_s[tid_block] = a_s[tid_block] + a_s[tid_block + s];
		__syncthreads();
	}

    if (tid_block == 0)
        a_in[blockIdx.x] = a_s[0];
}

int sum_array(int *a_in, int size)
{
    int sum = a_in[0];
    for(int i = 1 ; i < size ; i++)
        sum += a_in[i];
    return sum;
}

#ifdef OVERLAP
int overlaped_transfer_kernel(int factor, int stream_count)
{
    /* define and set variables */
	int *a_h, *a_d, *device_out_h;
	int sum_parralel, sum_seq;
    double seq_time, total_time, kernel_time;

    int size = 1024 * 1024 * factor;
    int block_size = 1024;
    int stream_size = size / stream_count;
    int block_count = (stream_size/block_size)/2;

    /* define and set kernel variables */
	dim3 grid_dim(block_count, 1, 1);
	dim3 block_dim(block_size, 1, 1);
    hipStream_t* streams = (hipStream_t *)malloc(sizeof(hipStream_t) * stream_count);
	for(int i=0;i<stream_count;i++)
        hipStreamCreate(&streams[i]);

    /* inital data on host */
    initialize_data_random_cudaMallocHost(&a_h, size);
    initialize_data_zero_cudaMallocHost(&device_out_h, block_count * stream_count);
    
    /* inital data on device */
    CUDA_CHECK_RETURN(hipMalloc((void **)&a_d, sizeof(int)*size));

    /* ### SEQUENTIAL ### */
    set_clock();
	sum_seq = sum_array(a_h, size);
    seq_time = get_elapsed_time();
        
    /* ### PARALLEL GPU ### */
	set_clock();

	int offset = 0, out_offset = 0;
	for(int stream_id=0 ; stream_id < stream_count ; stream_id++){
		hipMemcpyAsync(&a_d[offset], &a_h[offset], stream_size*sizeof(int), hipMemcpyHostToDevice, streams[stream_id]);
		add_kernel<<<grid_dim, block_dim, block_size*sizeof(int), streams[stream_id]>>>(&a_d[offset]);
		hipMemcpyAsync(&device_out_h[out_offset], &a_d[offset], block_count*sizeof(int), hipMemcpyDeviceToHost, streams[stream_id]);
		offset+=stream_size;
		out_offset+=block_count;
    }
    
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

    kernel_time = get_elapsed_time();
    set_clock();

	sum_parralel = sum_array(device_out_h, block_count * stream_count);

	total_time = get_elapsed_time();
	total_time += kernel_time;

    /* printing result and validation */
    printf("[TIME] Sequential: %.4f\n", seq_time);
	printf("[TIME] total parallel: %.4f\n", total_time);
    printf("[TIME] kernel_time : %.4f\n", kernel_time);
    printf("[SPEEDUP] sequentianal / parallel_time: %.4f\n", seq_time/total_time);
    printf("[VALIDATE] Parallel_sum: %d \tSeq_sum: %d\n", sum_parralel, sum_seq);
    printf("[VALIDATE] diffrentc of sums: %d\n", abs(sum_parralel - sum_seq));

    /* free alocated memory */
    free(streams);
    CUDA_CHECK_RETURN(hipHostFree(a_h));
    CUDA_CHECK_RETURN(hipHostFree(device_out_h));
    CUDA_CHECK_RETURN(hipFree(a_d));

    return 0;
}

int main(int argc, char * argv[])
{
    /* check and warning for user input */
    if(argc != 3){
		printf("Correct way to execute this program is:\n");
		printf("add_cuda factor(MB) stream_count\n");
		printf("For example:\nadd_cuda 40 4\n");
		return 1;
	}

    int factor = atoi(argv[1]);
    int stream_count = atoi(argv[2]);

    return overlaped_transfer_kernel(factor, stream_count);
}

#else
int main()
{
    return 0;
}
#endif