#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.cuh"
extern "C"{
    #include "fuzzy_timing.h"
    #include "helper_functions.h"
}

int sum_array(int *a_in, int size)
{
    int sum = a_in[0];
    for(int i = 1 ; i < size ; i++)
        sum += a_in[i];
    return sum;
}

#if defined(OVERLAP)
int overlaped_transfer_kernel(int factor, int stream_count)
{
    /* inform in array or out array operation */
    #if defined(IN_ARRAY)
    printf("[ARRAY] in array operation\n");
    #else
    printf("[ARRAY] out array operation\n");
    #endif

    /* define and set variables */
	int *a_h, *device_out_h;
	int sum_parralel, sum_seq;
    double seq_time, total_time, kernel_time;

    int size = 1024 * 1024 * factor;
    int block_size = 1024;
    int stream_size = size / stream_count;
    int block_count = (stream_size/block_size)/2;

    /* define and set kernel variables */
	dim3 grid_dim(block_count, 1, 1);
	dim3 block_dim(block_size, 1, 1);

    arguments* args = (arguments *)malloc(sizeof(arguments) * stream_count);
    hipStream_t* streams = (hipStream_t *)malloc(sizeof(hipStream_t) * stream_count);
	for(int i=0;i<stream_count;i++){
        hipStreamCreate(&streams[i]);

        /* inital data on device for each stream */
        CUDA_CHECK_RETURN(hipMalloc((void **)&args[i], sizeof(arguments)));
        CUDA_CHECK_RETURN(hipMalloc((void **)&(args[i].a_in), sizeof(int)*stream_size));

        #ifndef IN_ARRAY
        CUDA_CHECK_RETURN(hipMalloc((void **)&(args[i].out), sizeof(int) * block_count));    
        #endif
    }

    /* inital data on host */
    initialize_data_random_cudaMallocHost(&a_h, size);
    initialize_data_zero_cudaMallocHost(&device_out_h, block_count * stream_count);

    /* ### SEQUENTIAL ### */
    set_clock();
	sum_seq = sum_array(a_h, size);
    seq_time = get_elapsed_time();
        
    /* ### PARALLEL GPU ### */
	set_clock();

	int offset = 0, out_offset = 0;
	for(int stream_id=0 ; stream_id < stream_count ; stream_id++){
        hipMemcpyAsync(args[stream_id].a_in, &a_h[offset], stream_size*sizeof(int), hipMemcpyHostToDevice, streams[stream_id]);

        #ifdef IN_ARRAY
        add_kernel_in_array<<<grid_dim, block_dim, block_size*sizeof(int), streams[stream_id]>>>(args[stream_id]);
		hipMemcpyAsync(&device_out_h[out_offset], args[stream_id].a_in, block_count*sizeof(int), hipMemcpyDeviceToHost, streams[stream_id]);
        #else
		add_kernel<<<grid_dim, block_dim, block_size*sizeof(int), streams[stream_id]>>>(args[stream_id]);
		hipMemcpyAsync(&device_out_h[out_offset], args[stream_id].out, block_count*sizeof(int), hipMemcpyDeviceToHost, streams[stream_id]);
        #endif

        offset+=stream_size;
        out_offset+=block_count;
    }
    
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

    kernel_time = get_elapsed_time();
    set_clock();

	sum_parralel = sum_array(device_out_h, block_count * stream_count);

	total_time = get_elapsed_time();
	total_time += kernel_time;

    /* printing result and validation */
    printf("[TIME] Sequential: %.4f\n", seq_time);
	printf("[TIME] total parallel: %.4f\n", total_time);
    printf("[TIME] kernel_time : %.4f\n", kernel_time);
    printf("[SPEEDUP] sequentianal / parallel_time: %.4f\n", seq_time/total_time);
    printf("[VALIDATE] Parallel_sum: %d \tSeq_sum: %d\n", sum_parralel, sum_seq);
    printf("[VALIDATE] diffrentc of sums: %d\n", abs(sum_parralel - sum_seq));

    /* free alocated memory */
    for(int i=0;i<stream_count;i++){
        hipStreamDestroy(streams[i]);

        /* inital data on device for each stream */
        CUDA_CHECK_RETURN(hipFree(args[i].a_in));
        
        #ifndef IN_ARRAY
        CUDA_CHECK_RETURN(hipFree(args[i].out));
        #endif  
    }

    free(streams);
    free(args);
    CUDA_CHECK_RETURN(hipHostFree(a_h));
    CUDA_CHECK_RETURN(hipHostFree(device_out_h));

    return 0;
}

int main(int argc, char * argv[])
{
    printf("[MAIN] OVERLAP MAIN\n");

    /* check and warning for user input */
    if(argc != 3){
		printf("Correct way to execute this program is:\n");
		printf("add_cuda factor(MB) stream_count\n");
		printf("For example:\nadd_cuda 40 4\n");
		return 1;
	}

    int factor = atoi(argv[1]);
    int stream_count = atoi(argv[2]);

    return overlaped_transfer_kernel(factor, stream_count);
}

#elif defined(TEST)
int main()
{
    printf("[MAIN] TEST MAIN\n");

    set_clock();

    return 0;
}

#else
int one_add_kernel(int factor)
{
    /* inform in array or out array operation */
    #if defined(IN_ARRAY)
    printf("[ARRAY] in array operation\n");
    #else
    printf("[ARRAY] out array operation\n");
    #endif

    /* define and set variables */
	int *a_h, *device_out_h;
	int sum_parralel, sum_seq;
    double seq_time, total_time, kernel_time, mem_time;

    int size = 1024 * 1024 * factor;
    int block_size = 1024;
    int block_count = (size/block_size)/2;

    /* define and set kernel variables */
	dim3 grid_dim(block_count, 1, 1);
	dim3 block_dim(block_size, 1, 1);

    /* inital data on host */
    initialize_data_random(&a_h, size);
    initialize_data_zero(&device_out_h, block_count);
    
    /* inital data on device */
    arguments arg;
    CUDA_CHECK_RETURN(hipMalloc((void **)&arg, sizeof(arguments)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&(arg.a_in), sizeof(int) * size));

    #ifndef IN_ARRAY
    CUDA_CHECK_RETURN(hipMalloc((void **)&(arg.out), sizeof(int) * block_count));    
    #endif

    /* ### SEQUENTIAL ### */
    set_clock();
	sum_seq = sum_array(a_h, size);
    seq_time = get_elapsed_time();
        
    /* ### PARALLEL GPU ### */
	set_clock();

    hipMemcpy(arg.a_in, a_h, size*sizeof(int), hipMemcpyHostToDevice);

    mem_time = get_elapsed_time();
    set_clock();

    #ifdef IN_ARRAY
    add_kernel_in_array<<<grid_dim, block_dim, block_size*sizeof(int)>>>(arg);
    #else
    add_kernel<<<grid_dim, block_dim, block_size*sizeof(int)>>>(arg);
    #endif

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

    kernel_time = get_elapsed_time();
    set_clock();

    #ifdef IN_ARRAY
    hipMemcpy(device_out_h, arg.a_in, block_count*sizeof(int), hipMemcpyDeviceToHost);
    #else
    hipMemcpy(device_out_h, arg.out, block_count*sizeof(int), hipMemcpyDeviceToHost);
    #endif
    
    mem_time += get_elapsed_time();

    set_clock();

	sum_parralel = sum_array(device_out_h, block_count);

	kernel_time += get_elapsed_time();
	total_time = kernel_time + mem_time;

    /* printing result and validation */
    printf("[TIME] Sequential: %.4f\n", seq_time);
	printf("[TIME] total parallel: %.4f\n", total_time);
    printf("[TIME] kernel_time : %.4f\n", kernel_time);
    printf("[TIME] mem_time : %.4f\n", mem_time);
    printf("[SPEEDUP] sequentianal / parallel_time (total time): %.4f\n", seq_time/total_time);
    printf("[SPEEDUP] sequentianal / parallel_time (only operation): %.4f\n", seq_time/kernel_time);
    printf("[VALIDATE] Parallel_sum: %d \tSeq_sum: %d\n", sum_parralel, sum_seq);
    printf("[VALIDATE] diffrentc of sums: %d\n", abs(sum_parralel - sum_seq));

    /* free alocated memory */
    free(a_h);
    free(device_out_h);

    CUDA_CHECK_RETURN(hipFree(arg.a_in));

    #ifndef IN_ARRAY
    CUDA_CHECK_RETURN(hipFree(arg.out));
    #endif

    return 0;
}

int main(int argc, char * argv[])
{
    printf("[MAIN] else MAIN (not overlap transfer)\n");
    
    /* check and warning for user input */
    if(argc != 2){
		printf("Correct way to execute this program is:\n");
		printf("add_cuda factor(MB)\n");
		printf("For example:\nadd_cuda 40\n");
		return 1;
	}

    int factor = atoi(argv[1]);

    return one_add_kernel(factor);
}
#endif